#include "hip/hip_runtime.h"
/**
 * @file conv_kernel.cu
 * @brief CUDA implementation of 2D convolutional layer forward pass
 *
 * This file contains the CUDA kernel implementation for the forward pass
 * of 2D convolutional layers, using shared memory for efficient computation.
 */

#include "kernels.h"

#define BLOCK_SIZE 256
#define TILE_SIZE 16

/**
 * @brief CUDA kernel for 2D convolution forward pass with single sample
 *
 * This kernel implements the forward pass of a 2D convolutional layer using
 * shared memory tiling for improved performance. It supports:
 * - Multiple input and output channels
 * - Configurable kernel size
 * - Zero padding
 * - Optional bias
 *
 * @param input Input feature map [C_in, H_in, W_in]
 * @param weights Weight tensor [C_out, C_in, kernel_size, kernel_size]
 * @param bias Bias tensor [C_out] (can be nullptr if no bias)
 * @param output Output feature map [C_out, H_out, W_out]
 * @param C_in Number of input channels
 * @param C_out Number of output channels
 * @param H_in Input height
 * @param W_in Input width
 * @param H_out Output height
 * @param W_out Output width
 * @param kernel_size Size of the convolution kernel (assumed square)
 * @param padding Padding size
 */
__global__ void conv_forward_kernel_batch_dim_1(
    const real_t *input, // Input feature map [C_in, H_in, W_in]
    const real_t
        *weights,       // Weight tensor [C_out, C_in, kernel_size, kernel_size]
    const real_t *bias, // Bias tensor [C_out] (can be nullptr if no bias)
    real_t *output,     // Output feature map [C_out, H_out, W_out]
    size_t C_in,        // Input channels
    size_t C_out,       // Output channels
    size_t H_in, size_t W_in,   // Input height and width
    size_t H_out, size_t W_out, // Output height and width
    size_t kernel_size,         // Kernel size (assuming square kernel)
    size_t padding              // Padding size
) {
    // Thread and block indices
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Calculate output position for this thread
    int out_x = bx * TILE_SIZE + tx;
    int out_y = by * TILE_SIZE + ty;

    // Early exit if outside output bounds
    if (out_x >= W_out || out_y >= H_out)
        return;

    // Shared memory for input tile
    // Size needs to accommodate kernel overlap: TILE_SIZE + kernel_size - 1
    int SHARED_SIZE = TILE_SIZE + kernel_size - 1;
    extern __shared__ real_t shared_input[];

    // Process each output channel
    for (int c_out = 0; c_out < C_out; c_out++) {
        real_t result = 0.0f;

        // Process each input channel
        for (int c_in = 0; c_in < C_in; c_in++) {
            // Load input data into shared memory with padding handling
            // Each thread loads multiple elements to fill the shared memory
            // tile
            for (int load_y = ty; load_y < SHARED_SIZE; load_y += blockDim.y) {
                for (int load_x = tx; load_x < SHARED_SIZE;
                     load_x += blockDim.x) {
                    // Calculate input coordinates with padding offset
                    int in_x = bx * TILE_SIZE + load_x - padding;
                    int in_y = by * TILE_SIZE + load_y - padding;

                    // Handle boundary conditions (zero-padding)
                    if (in_x >= 0 && in_x < W_in && in_y >= 0 && in_y < H_in) {
                        int input_idx = c_in * H_in * W_in + in_y * W_in + in_x;
                        shared_input[load_y * SHARED_SIZE + load_x] =
                            input[input_idx];
                    } else {
                        shared_input[load_y * SHARED_SIZE + load_x] =
                            0.0f; // Zero padding
                    }
                }
            }

            // Synchronize to ensure all threads have loaded their data
            __syncthreads();

            // Perform convolution using shared memory
            for (int ky = 0; ky < kernel_size; ky++) {
                for (int kx = 0; kx < kernel_size; kx++) {
                    // Index into shared memory (accounting for padding offset)
                    int shared_x = tx + kx;
                    int shared_y = ty + ky;

                    // Weight index: [c_out, c_in, ky, kx]
                    int weight_idx = c_out * C_in * kernel_size * kernel_size +
                                     c_in * kernel_size * kernel_size +
                                     ky * kernel_size + kx;

                    // Accumulate the convolution result
                    result += shared_input[shared_y * SHARED_SIZE + shared_x] *
                              weights[weight_idx];
                }
            }

            // Synchronize before loading next input channel
            __syncthreads();
        }

        // Add bias if provided
        if (bias != nullptr) {
            result += bias[c_out];
        }

        // Store result to output
        int output_idx = c_out * H_out * W_out + out_y * W_out + out_x;
        output[output_idx] = result;
    }
}

/**
 * @brief Calculate output dimensions for 2D convolution
 *
 * Computes the output dimensions of a 2D convolution operation given
 * input dimensions, kernel size, and padding.
 *
 * @param H_in Input height
 * @param W_in Input width
 * @param kernel_size Size of the convolution kernel
 * @param padding Padding size
 * @param H_out [out] Output height
 * @param W_out [out] Output width
 */
void calculate_output_dimensions(size_t H_in, size_t W_in, size_t kernel_size,
                                 size_t padding, size_t &H_out, size_t &W_out) {
    // for stride = 1:
    H_out = H_in + 2 * padding - kernel_size + 1;
    W_out = W_in + 2 * padding - kernel_size + 1;
}

/**
 * @brief Host function to launch 2D convolution forward pass
 *
 * This function handles the CUDA kernel launch for the 2D convolutional layer
 * forward pass. Currently only supports single sample processing.
 *
 * @param input Input feature map
 * @param weights Convolution weights
 * @param bias Bias vector (can be nullptr)
 * @param output Output feature map
 * @param batch_size Number of samples in the batch (must be 1)
 * @param input_channels Number of input channels
 * @param output_channels Number of output channels
 * @param kernel_h Height of the convolution kernel
 * @param kernel_w Width of the convolution kernel
 * @param H_in Input height
 * @param W_in Input width
 * @throws std::runtime_error if batch_size > 1 or kernel launch fails
 */
void conv2d_forward(const real_t *input, const real_t *weights,
                    const real_t *bias, real_t *output, size_t batch_size,
                    size_t input_channels, size_t output_channels,
                    size_t kernel_h, size_t kernel_w, size_t H_in,
                    size_t W_in) {

    size_t padding = 1; // TODO: implement padding
    size_t H_out, W_out;
    calculate_output_dimensions(H_in, W_in, kernel_h, padding, H_out, W_out);

    dim3 blockDim(TILE_SIZE, TILE_SIZE);
    dim3 gridDim((W_out + TILE_SIZE - 1) / TILE_SIZE,
                 (H_out + TILE_SIZE - 1) / TILE_SIZE);

    int SHARED_SIZE = TILE_SIZE + kernel_h - 1;
    size_t shared_mem_size = SHARED_SIZE * SHARED_SIZE * sizeof(real_t);

    printf("\n------ Conv2D Forward ------\n");

    if (batch_size == 1) {
        conv_forward_kernel_batch_dim_1<<<gridDim, blockDim, shared_mem_size>>>(
            input, weights, bias, output, input_channels, output_channels, H_in,
            W_in, H_out, W_out, kernel_h, padding);
    } else {
        throw std::runtime_error(
            "Batch size > 1 not implemented for conv2d_forward");
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Conv2D kernel launch error: %s\n",
               hipGetErrorString(err));
    }
}
